#include "hip/hip_runtime.h"
#include "nbody.h"

__global__ void coulombGaForce(Ga *p1,Ele* p2, float dt, int n_ga,int n_ele)//Ga库仑力计算
{
    double k = 8.988e9;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n_ga)
    {
        double ax = 0.0;
        double ay = 0.0;
        double az = 0.0;
        for(int j=0;j<n_ga;j++)//离子与离子
        {
            if(j==i)
                continue;
            double dx = p1[i].x - p1[j].x;
            double dy = p1[i].y - p1[j].y;
            double dz = p1[i].z - p1[j].z;
            double distSqr = dx * dx + dy * dy + dz * dz;
            double invDist = rsqrtf(distSqr);//平方根的倒数，rsqrtf(25) = 0.2
            double invDist3 = invDist * invDist * invDist;
            ax += k * p1[i].charge * p1[j].charge* dx * invDist3;
            ay += k * p1[i].charge * p1[j].charge* dy * invDist3;
            az += k * p1[i].charge * p1[j].charge* dz * invDist3;
        }
        for(int j=0;j<n_ele;j++)//离子与电子
        {
            double dx = p1[i].x - p2[j].x;
            double dy = p1[i].y - p2[j].y;
            double dz = p1[i].z - p2[j].z;
            double distSqr = dx * dx + dy * dy + dz * dz;
            double invDist = rsqrtf(distSqr);//平方根的倒数，rsqrtf(25) = 0.2
            double invDist3 = invDist * invDist * invDist;

            ax += k * p1[i].charge * p2[j].charge* dx * invDist3;   //库仑力
            ay += k * p1[i].charge * p2[j].charge* dy * invDist3;
            az += k * p1[i].charge * p2[j].charge* dz * invDist3;
        }
        p1[i].vx += dt * ax / p1[i].mass;   //v=v0+at=v0+F/m*t
        p1[i].vy += dt * ay / p1[i].mass;
        p1[i].vz += dt * az / p1[i].mass;
    }
}

__global__ void coulombEleForce(Ga *p1,Ele* p2, float dt, int n_ga,int n_ele)//Ele 库仑力计算
{
    double k = 8.988e9;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n_ele)
    {
        double ax = 0.0;
        double ay = 0.0;
        double az = 0.0;
        for(int j=0;j<n_ga;j++)//电子与离子
        {
            double dx = p2[i].x - p1[j].x;
            double dy = p2[i].y - p1[j].y;
            double dz = p2[i].z - p1[j].z;
            double distSqr = dx * dx + dy * dy + dz * dz;
            double invDist = rsqrtf(distSqr);//平方根的倒数，rsqrtf(25) = 0.2
            double invDist3 = invDist * invDist * invDist;

            ax += k * p2[i].charge * p1[j].charge* dx * invDist3;
            ay += k * p2[i].charge * p1[j].charge* dy * invDist3;
            az += k * p2[i].charge * p1[j].charge* dz * invDist3;
        }
        for(int j=0;j<n_ele;j++) //电子与电子
        {
            if(j==i)
                continue;
            double dx = p2[i].x - p2[j].x;
            double dy = p2[i].y - p2[j].y;
            double dz = p2[i].z - p2[j].z;
            double distSqr = dx * dx + dy * dy + dz * dz;
            double invDist = rsqrtf(distSqr);//平方根的倒数，rsqrtf(25) = 0.2
            double invDist3 = invDist * invDist * invDist;

            ax += k * p2[i].charge * p2[j].charge * dx * invDist3;
            ay += k * p2[i].charge * p2[j].charge * dy * invDist3;
            az += k * p2[i].charge * p2[j].charge * dz * invDist3;
        }
        p2[i].vx += dt * ax / p2[i].mass;
        p2[i].vy += dt * ay / p2[i].mass;
        p2[i].vz += dt * az / p2[i].mass;
    }
}

__global__ void integrate_ga_position(Ga *p, float dt, int n_ga,double thrZ)//Ga位置更新
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n_ga)
    {
        p[i].x += p[i].vx * dt;  //x=x0+v*t
        p[i].y += p[i].vy * dt;
        p[i].z += p[i].vz * dt;

        /*新加代码，在离焦平面z_height距离时假设中和掉，不再受库仑力作用*/
        //double z_height = 0.02;//thrZ = 0.159354, start z = 0.1525
        // double z_height = 0.005;
        // //double z_height = 0.003;
        // //double z_height = 0.007;
        // //double z_height = 0.0005;
        // if((p[i].z+z_height) >= thrZ)
        // {
        //     p[i].charge = 0;
        // }
        // /*新加代码结束*/

        if(p[i].z > thrZ)
        {
            p[i].charge = 0;
            p[i].vx = 0;
            p[i].vy = 0;
            p[i].vz = 0;
        }
    }
}

__global__ void integrate_ele_position(Ele* p, float dt, int n_ele,double thrZ)//Ele位置更新
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n_ele)
    {
        p[i].x += p[i].vx * dt;
        p[i].y += p[i].vy * dt;
        p[i].z += p[i].vz * dt;
        if(p[i].z > thrZ)
        {
            p[i].charge = 0;
            p[i].vx = 0;
            p[i].vy = 0;
            p[i].vz = 0;
        }
    }
}
