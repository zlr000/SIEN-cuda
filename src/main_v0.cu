#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include "particle.h"
#include "nbody.h"
#include <fstream>
#include <iostream>
#include "settings.h"

int main()
{
    std::queue<long> gaTimes;
    std::queue<Ga> gaData; 
    std::queue<long> eleTimes;
    std::queue<Ele> eleData;
    //std::vector<std::vector<Ga>> gaData = parsing_ga_file(gaFile,numDelLine,N_p_ga,initZ_ga);
    //std::vector<std::vector<Ele>> eleData = parsing_ele_file(eleFile,numDelLine,N_p_ele,initZ_ga,initZ_ele,theta);
    gaData = parsing_ga_file(gaTimes);
    eleData = parsing_ele_file(eleTimes);
    std::cout << "Ga number:" << gaData.size() << " " << "Ga times number:" << gaTimes.size() << std::endl;
    std::cout << "Ele number:" << eleData.size() << " " << "Ele times number:" << eleTimes.size() << std::endl;
    //std::cout << "Ga  bunch number:"<<gaData.size()  << " " <<"Particle number per bunch:"<< gaData[0].size()  << std::endl;
    //std::cout << "ele bunch number:"<<eleData.size() << " " <<"Particle number per bunch:"<< eleData[0].size() << std::endl;
    std::vector<Ga> gaBuff;
    std::vector<Ele> eleBuff;
    std::vector<Ga> focalPlaneGa;
    std::vector<Ele> focalPlaneEle;
    std::vector<Ga> zhGa;
    std::vector<Ele> zhEle;
    int gaNum = 0,eleNum = 0;//ga,ele数
    long iterNum = (t_total + dt / 2.0) / dt;
    long focalStep = (dt_focal  + dt / 2.0)/dt;
    long  focalStart = t_focal_start / dt;
    long  log_step = (iterNum + dt / 2.0) / 100;
    //long ele_offset_step = 0;
    //std::cout << "Total/ga/ele iters:"<< iterNum << " " << gaStep << " " << eleStep << std::endl;
    Ga* ga_gpu;
    Ele* ele_gpu;
    StartTimer();
    std::cout << iterNum << std::endl;
    for(long i = 0; i < iterNum;i++)
    {
        //每次log输出GaBUff/GaFocal/EleBuff/EleFocal数量信息
        //每次log保存一次GaBUff/GaFocal/EleBuff/EleFocal的中间结果信息
        if(i%log_step==0)
        {
            //输出log信息
            const double tElapsed = GetTimer() / 1000.0;
            StartTimer();
            std::cout << "Iter number:" << i << "/" << iterNum << " Total time:" << tElapsed << std::endl;
            std::cout << "GaBUff/GaFocal/EleBuff/EleFocal number:";
            std::cout << gaBuff.size() << " "<< focalPlaneGa.size() << " "<< eleBuff.size() << " " << focalPlaneEle.size() << std::endl;
            std::cout << "neutralization number Ga/Ele:" << zhGa.size() << " " << zhEle.size() << std::endl;
            //保存中间文件
            string suffix = "_" + std::to_string((int)(i/log_step)) + ".txt";
            writeFile<Ga>(positionPathGaPrefix+suffix,gaBuff);
            writeFile<Ga>(focalPlanePathGaPrefix+suffix,focalPlaneGa);
            writeFile<Ele>(positionPathElePrefix+suffix,eleBuff);
            writeFile<Ele>(focalPlanePathElePrefix+suffix,focalPlaneEle); 
        }

        //在i为10的整数倍，判断是否需要添加ga离子
        //if(i % 10 == 0)
        if(true)
        {
            //ga离子判断
            //if(gaTimes[0] <= i)//增加ga离子
            if(gaTimes.front() <= i)
            {
                if(!gaBuff.empty())//如果ga离子非空，释放显存
                {
                    //hipDeviceSynchronize();
                    int bytes_ga = gaNum * sizeof(Ga);
                    hipMemcpy(gaBuff.data(),ga_gpu,bytes_ga,hipMemcpyDeviceToHost);
                    hipFree(ga_gpu);
                }
                gaBuff.push_back(gaData.front());
                gaData.pop();
                gaTimes.pop();
                gaNum = gaBuff.size();
                int bytes_ga = gaNum * sizeof(Ga);
                hipMallocManaged(&ga_gpu, bytes_ga);
                hipMemcpy(ga_gpu, gaBuff.data(), bytes_ga, hipMemcpyHostToDevice);
            }
            //ele离子判断,添加电子
            /*
            if(eleTimes.front() <= i)
            {
                if(!eleBuff.empty())
                {
                    int bytes_ele = eleNum * sizeof(Ele);
                    hipMemcpy(eleBuff.data(),ele_gpu,bytes_ele,hipMemcpyDeviceToHost);
                    hipFree(ele_gpu);
                }
                eleBuff.push_back(eleData.front());
                eleData.pop();
                eleTimes.pop();
                eleNum = eleBuff.size();
                int bytes_ele = eleNum * sizeof(Ele);
                hipMallocManaged(&ele_gpu,bytes_ele);
                hipMemcpy(ele_gpu,eleBuff.data(),bytes_ele,hipMemcpyHostToDevice);
            }
            */
        }

        size_t threadsPerBlock = BLOCK_SIZE;

        //size_t threadsPerBlockGa = N_p_ga;
        //size_t threadsPerBlockEle = N_p_ele;
        size_t numberOfBlocksGa = (gaNum + threadsPerBlock - 1)/threadsPerBlock;
        size_t numberOfBlocksEle =(eleNum+ threadsPerBlock - 1)/threadsPerBlock;
        coulombGaForce<<<numberOfBlocksGa,threadsPerBlock>>>(ga_gpu,ele_gpu, dt,gaNum,eleNum);
        coulombEleForce<<<numberOfBlocksEle,threadsPerBlock>>>(ga_gpu,ele_gpu, dt,gaNum,eleNum);
        integrate_ga_position<<<numberOfBlocksGa,threadsPerBlock>>>(ga_gpu,dt,gaNum);
        integrate_ele_position<<<numberOfBlocksEle,threadsPerBlock>>>(ele_gpu,dt,eleNum);      
        if(i % focalStep == 0 && i >= focalStart)
        {
            int bytes_ga = gaNum * sizeof(Ga);
            int bytes_ele = eleNum * sizeof(Ele);
            hipMemcpy(gaBuff.data(),ga_gpu,bytes_ga,hipMemcpyDeviceToHost);//
            hipMemcpy(eleBuff.data(),ele_gpu,bytes_ele,hipMemcpyDeviceToHost);//
            //S2
            std::vector<Ga> tmpGaBuff;
            std::vector<Ele> tmpEleBuff;
            //首先判断中和，gpt距离为0.529A = 5.29 * 1e-11m
            //double zhonghe = 5.29 * 1e-11;
            double zhonghe = 5.29 * 1e-5;
	        //double zhonghe = 5.29 * 1e-4;
            //for(int j=gaBuff.size()-1;j>=0;j--)
            // double z_thr = 156 * 1e-3;//z阈值
            // for(size_t j=0;j<gaBuff.size();j++)
            // {
            //     if(gaBuff[j].z > z_thr)
            //     {
            //         gaBuff[j].charge = 0;
            //     }
            // }
            // for(size_t j=0;j<gaBuff.size();j++)
            // {
            //         gaBuff[j].charge = 0;
            // }

            // for(size_t j=0;j<gaBuff.size();j++)
            // {
            //     //for(int k=eleBuff.size()-1;k>=0;k--)
            //     for(size_t k = 0;k<eleBuff.size();k++)
            //     {
            //         double x2 = (gaBuff[j].x - eleBuff[k].x) * (gaBuff[j].x - eleBuff[k].x);
            //         double y2 = (gaBuff[j].y - eleBuff[k].y) * (gaBuff[j].y - eleBuff[k].y);
            //         double z2 = (gaBuff[j].z - eleBuff[k].z) * (gaBuff[j].z - eleBuff[k].z);
            //         double r2 = x2 + y2 + z2;
            //         double r  = sqrt(r2);
            //         if(r < zhonghe && gaBuff[j].charge > 1e-19 && eleBuff[k].charge < -1e-19)//1.6e-19
            //         {
            //             gaBuff[j].charge = 0;
            //             eleBuff[k].charge = 0;
            //             zhGa.push_back(gaBuff[j]);
            //             zhEle.push_back(eleBuff[k]);
			//             //gaBuff.erase(gaBuff.begin() + j);
            //             //eleBuff.erase(eleBuff.begin() + k);
            //             break;
            //         }
            //     }
            // }


            //然后判断到达焦平面
            for(size_t j=0;j<gaBuff.size();j++)
            {
                if(gaBuff[j].z >= S2 * 1e-3)
                {
                    focalPlaneGa.push_back(gaBuff[j]);
                }
                else
                {
                    tmpGaBuff.push_back(gaBuff[j]);
                }
            }
            for(size_t j=0;j<eleBuff.size();j++)
            {
                if(eleBuff[j].z < S2 * 1e-3)
                {
                    tmpEleBuff.push_back(eleBuff[j]);
                }
                else
                {
                    focalPlaneEle.push_back(eleBuff[j]);
                }
                // else
                // {
                //     focalPlaneEle.push_back(eleBuff[j]);
                //     //tmpEleBuff.push_back(eleBuff[j]);
                // }
            }
            gaBuff = tmpGaBuff;
            eleBuff = tmpEleBuff;
            hipFree(ga_gpu);
            hipFree(ele_gpu);
            gaNum = gaBuff.size();
            eleNum = eleBuff.size();
            bytes_ga = gaNum * sizeof(Ga);
            bytes_ele = eleNum * sizeof(Ele);
            hipMallocManaged(&ga_gpu, bytes_ga);
            hipMallocManaged(&ele_gpu, bytes_ele);
            hipMemcpy(ga_gpu, gaBuff.data(), bytes_ga, hipMemcpyHostToDevice);
            hipMemcpy(ele_gpu,eleBuff.data(),bytes_ele,hipMemcpyHostToDevice);
            // hipError_t err = hipGetLastError();
            // if (err != hipSuccess) {
            //     printf("CUDA Error: %s\n", hipGetErrorString(err));
            //     // Possibly: exit(-1) if program cannot continue....
            // }

        }
        //if((i+1)%gaStep == 0 || (i+1)%eleStep ==0)
        //{
        //    hipDeviceSynchronize();
        //}
    } 

    //保存位置文件
    std::cout << "Final output:" << gaBuff.size() << " "<< focalPlaneGa.size() << " "<< eleBuff.size() << " " << focalPlaneEle.size() << std::endl;
    //writeFile(positionPathGa,gaBuff);
    writeFile<Ga>(positionPathGa,gaBuff);
    writeFile<Ga>(focalPlanePathGa,focalPlaneGa);
    writeFile<Ele>(positionPathEle,eleBuff);
    writeFile<Ele>(focalPlanePathEle,focalPlaneEle);

    return 0;
}
