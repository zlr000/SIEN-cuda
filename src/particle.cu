#include "hip/hip_runtime.h"
#include "particle.h"
#include<fstream>
#include<iostream>
#include<math.h>
#include<settings.h>
#include <algorithm>
#include <numeric>

int CountLines(std::string& filename);

queue<Ga> parsing_ga_file(queue<long> &gaTimes)  //解析Ga文件
{
    int n1 = CountLines(gaFileD) - numDelLine;//总行数减去删掉行
    int cols = 8;

    std::ifstream file;
    file.open(gaFileD, std::ios::in);
    if (!file.is_open())
    {
        std::cout << "Fail to open file" << std::endl;
        exit(-1);
    }
    for (int i = 0; i < numDelLine; i++)
    {
        std::string tmpLine;
        std::getline(file, tmpLine);
    }
    std::vector<std::vector<double> > tmp(n1, std::vector<double>(cols));
    for (int i = 0; i < n1; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            file >> tmp[i][j];
        }
    }
    file.close();
    vector<Ga> gaData_(n1);
    for(int i = 0;i<n1;i++)
    {
        Ga tmpGa;
        tmpGa.id = tmp[i][0];
        tmpGa.vz = sqrt((2 * 1.6e-19 * tmp[i][6]) / tmpGa.mass);
        tmpGa.vx = tmp[i][3] * tmpGa.vz;
        tmpGa.vy = tmp[i][4] * tmpGa.vz;
        tmpGa.x  = tmp[i][1] * 1e-6;
        tmpGa.y  = tmp[i][2] * 1e-6;
        tmpGa.z  = initZ_ga * 1e-3;
        gaData_[i] = tmpGa;
    }

    int n2 = CountLines(gaFileT) - numDelLine;//总行数减去删掉行
    vector<long> gaTimes_(n2);
    //gaTimes_ = vector<long>(n2);
    file.open(gaFileT, std::ios::in);
    for (int i = 0; i < numDelLine; i++)
    {
        std::string tmpLine;
        std::getline(file, tmpLine);
    }
    //tmp n2
    for (int i = 0; i < n2; i++)
    {
        string tmpS;
        double tmpD;
        //13按Time started时间，14按Time Removed时间
        //13对应3，14对应2
        for (int j = 0; j < 14; j++)
        {
            file >> tmpS;
        }
        file >> tmpD;
        //tmpD = tmpD * 1e-12;
        gaTimes_[i] = (tmpD * 1e-12) / dt;
        //gaTimes_[i] = (gaTimes_[i] + 4) / 10 * 10;//此公式保证gaTimes数据为10的倍数
        for(int j=0;j<2;j++)
        {
            file >> tmpS;
        }
    }
    file.close();
    //判断长度是否一致，不一致则给出报错信息并退出程序
    if((gaData_.size() != gaTimes_.size()) ||(gaData_.size() == 0))
    {
        cout << "The size of gaData should bigger than 0 and equal to gaTimes!"<<endl;
        exit(-1);
    }
    //gaData 排序
    std::vector<int> indices(gaData_.size());//初始化索引数组
    std::iota(indices.begin(), indices.end(), 0);//生成从0开始连续递增的数组
    std::sort(indices.begin(), indices.end(), [&gaTimes_](size_t i1, size_t i2) { return gaTimes_[i1] < gaTimes_[i2]; });
    
    //vector<Ga> gaData(n1);
    queue<Ga> gaData;
    //gaTimes = vector<long>(n1);
    for(int i=0;i<n1;i++)
    {
        gaData.push(gaData_[indices[i]]);
        gaTimes.push(gaTimes_[indices[i]] - gaTimes_[indices[0]]);
    }
    
    //以下代码为输出实验
    // for(int i=0;i<101;i++)
    // {
    //     std::cout << gaData[i].id << " " << indices[i] << " " << gaTimes_[i] << std::endl;
    // }
    // //std::sort(gaTimes.begin(), gaTimes.end());
    // cout << endl;
    // for(int i=0;i<101;i++)
    // {
    //     std::cout << gaData[i].id << " " << gaTimes[i] << std::endl;
    // }
    // std::cout << "end" << std::endl;

    return gaData;
}

//vector<Ele> parsing_ele_file(vector<long> &eleTimes)
queue<Ele> parsing_ele_file(queue<long> &eleTimes)
{
    //theta 已转为弧度制
    double C1 = sin(theta) * S1;
    double C2 = S2 - cos(theta) * S1;
    C1 = C1 * 1e-3;
    C2 = C2 * 1e-3;

    int n1 = CountLines(eleFileD) - numDelLine;
    int cols = 8;
    std::ifstream file;
    file.open(eleFileD, std::ios::in);
    if (!file.is_open())
    {
        std::cout << "Fail to open file" << std::endl;
        exit(-1);
    }
    for (int i = 0; i < numDelLine; i++)
    {
        std::string tmpLine;
        std::getline(file, tmpLine);
    }
    std::vector<std::vector<double> > tmp(n1, std::vector<double>(cols));
    for (int i = 0; i < n1; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            file >> tmp[i][j];
        }
    }
    file.close();
    vector<Ele> eleData_(n1);
    for(int i = 0;i<n1;i++)
    {
        Ele tmpEle;
        tmpEle.id = tmp[i][0];
        tmpEle.vz = sqrt((2 * 1.6e-19 * tmp[i][6]) / tmpEle.mass);
        tmpEle.vx = tmp[i][3] * tmpEle.vz;
        tmpEle.vy = tmp[i][4] * tmpEle.vz;
        tmpEle.x  = tmp[i][1] * 1e-6;
        tmpEle.y  = tmp[i][2] * 1e-6;
        tmpEle.z  = initZ_ele * 1e-3;

        //坐标变换
        double tmpX,tmpZ;
        double tmpVX,tmpVZ;
        tmpX  = cos(theta) * tmpEle.x -  sin(theta) * tmpEle.z + C1;
        tmpZ  = sin(theta) * tmpEle.x +  cos(theta) * tmpEle.z + C2;
        tmpVX = cos(theta) * tmpEle.vx - sin(theta) * tmpEle.vz;
        tmpVZ = sin(theta) * tmpEle.vx + cos(theta) * tmpEle.vz;
        tmpEle.x = tmpX;
        tmpEle.z = tmpZ-0.004;
        tmpEle.vx= tmpVX;
        tmpEle.vz= tmpVZ;

        eleData_[i] = tmpEle;        
    }
    int n2 = CountLines(eleFileT) - numDelLine;
    vector<long> eleTimes_(n2);
    file.open(eleFileT,std::ios::in);
    for (int i = 0; i < numDelLine; i++)
    {
        std::string tmpLine;
        std::getline(file, tmpLine);
    }
    for(int i=0;i<n2;i++)
    {
        string tmpS;
        double tmpD;
        //13改成14，按Time Removed时间
        for (int j = 0; j < 14; j++)
        {
            file >> tmpS;
        }
        file >> tmpD;
        eleTimes_[i] = (tmpD * 1e-12) / dt;
        //eleTimes_[i] = (eleTimes_[i] + 4) / 10 * 10;
        for(int j=0;j<2;j++)
        {
            file >> tmpS;
        }        
    }
    file.close();
    //判断长度是否一致，不一致则给出报错信息并退出程序
    if((eleData_.size() != eleTimes_.size()) ||(eleData_.size() == 0))
    {
        cout << "The size of eleData should bigger than 0 and equal to eleTimes!"<<endl;
        exit(-1);
    }
    std::vector<int> indices(eleData_.size());//初始化索引数组
    std::iota(indices.begin(), indices.end(), 0);//生成从0开始连续递增的数组
    std::sort(indices.begin(), indices.end(), [&eleTimes_](size_t i1, size_t i2) { return eleTimes_[i1] < eleTimes_[i2]; });

    //vector<Ele> eleData(n1);
    //eleTimes = vector<long>(n1);
    queue<Ele> eleData;
    long dt_int = (long)((dt_ele + dt / 2) / dt);
    for(int i=0;i<n1;i++)
    {
        eleData.push(eleData_[indices[i]]);

        //case1 按照文件时间发射
        eleTimes.push(eleTimes_[indices[i]] - eleTimes_[indices[0]]);
        //case1 结束

        //case2 按照自定义dt_ele发射
        //eleTimes.push(dt_int);
        //dt_int += (long)((dt_ele + dt / 2) / dt);
        //case2 结束

    }
    long iterNum = (t_total + dt / 2.0) / dt;
    while(eleTimes.back() < iterNum)
    {
        long addOffset = eleTimes.back();
        for(int i=0;i<n1;i++)
        {
            eleData.push(eleData_[indices[i]]);
            eleTimes.push(eleTimes_[indices[i]] - eleTimes_[indices[0]] + addOffset);
        }

    }
    return eleData;
}

int CountLines(std::string& filename)
{
    std::ifstream Readfile;
    int line = 0;
    std::string tmp;

    Readfile.open(filename, std::ios::in);
    if (!Readfile.is_open())
    {
        std::cout << "cannot open file" << filename << std::endl;
        return -1;
    }
    else
    {
        while (std::getline(Readfile, tmp, '\n'))
        {
            line++;
        }
        Readfile.close();
        return line;
    }
}

// template <typename T>
// void writeFile(string filename,vector<T> data)
// {
//     std::ofstream fout;
//     fout.open(filename, std::ios::out);
//     if (!fout.is_open())
//     {
//         std::cout << filename << " cannot open!" << std::endl;
//         exit(-1);
//     }
//     for(int i=0;i<data.size();i++)
//     {
//         fout << data[i].id << " ";
//         fout << data[i].x << " ";
//         fout << data[i].y << " ";
//         fout << data[i].z;
//         fout << std::endl;
//     }
//     fout.close();
// }
