#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include "particle.h"
#include "nbody.h"
#include <fstream>
#include <iostream>
#include "settings.h"
#include <algorithm>

int main()
{
    std::queue<long> gaTimes;
    std::queue<Ga> gaData; 
    std::queue<long> eleTimes;
    std::queue<Ele> eleData;
    //std::vector<std::vector<Ga>> gaData = parsing_ga_file(gaFile,numDelLine,N_p_ga,initZ_ga);
    //std::vector<std::vector<Ele>> eleData = parsing_ele_file(eleFile,numDelLine,N_p_ele,initZ_ga,initZ_ele,theta);
    gaData = parsing_ga_file(gaTimes);
    eleData = parsing_ele_file(eleTimes);
    std::cout << "Ga number:" << gaData.size() << " " << "Ga times number:" << gaTimes.size() << std::endl;
    std::cout << "Ele number:" << eleData.size() << " " << "Ele times number:" << eleTimes.size() << std::endl;
    //std::cout << "Ga  bunch number:"<<gaData.size()  << " " <<"Particle number per bunch:"<< gaData[0].size()  << std::endl;
    //std::cout << "ele bunch number:"<<eleData.size() << " " <<"Particle number per bunch:"<< eleData[0].size() << std::endl;
    std::vector<Ga> gaBuff;
    std::vector<Ele> eleBuff;
    std::vector<Ga> focalPlaneGa;
    std::vector<Ele> focalPlaneEle;
    std::vector<Ga> zhGa;
    std::vector<Ele> zhEle;
    int gaNum = 0,eleNum = 0;//ga,ele数
    long iterNum = (t_total + dt / 2.0) / dt; 
    long focalStep = (dt_focal  + dt / 2.0)/dt;
    //long  focalStart = t_focal_start / dt;
    long  log_step = (iterNum + dt / 2.0) / 100;
    //long ele_offset_step = 0;
    //std::cout << "Total/ga/ele iters:"<< iterNum << " " << gaStep << " " << eleStep << std::endl;
    Ga* ga_gpu;
    Ele* ele_gpu;
    hipMallocManaged(&ga_gpu, 10000 * sizeof(Ga));
    hipMallocManaged(&ele_gpu,1000 * sizeof(Ele));
    StartTimer();
    std::cout << iterNum << std::endl;
    for(long i = 0; i < iterNum;i++)
    {
        //每次log输出GaBUff/GaFocal/EleBuff/EleFocal数量信息
        //每次log保存一次GaBUff/GaFocal/EleBuff/EleFocal的中间结果信息
        if(i%log_step==0)
        {
            //输出log信息
            const double tElapsed = GetTimer() / 1000.0;
            StartTimer();
            std::cout << "Iter number:" << i << "/" << iterNum << " Total time:" << tElapsed << std::endl;
            std::cout << "GaBUff/GaFocal/EleBuff/EleFocal number:";
            std::cout << gaBuff.size() << " "<< focalPlaneGa.size() << " "<< eleBuff.size() << " " << focalPlaneEle.size() << std::endl;
            std::cout << "neutralization number Ga/Ele:" << zhGa.size() << " " << zhEle.size() << std::endl;
            //保存中间文件
            string suffix = "_" + std::to_string((int)(i/log_step)) + ".txt";
            writeFile<Ga>(positionPathGaPrefix+suffix,gaBuff);
            writeFile<Ga>(focalPlanePathGaPrefix+suffix,focalPlaneGa);
            writeFile<Ele>(positionPathElePrefix+suffix,eleBuff);
            writeFile<Ele>(focalPlanePathElePrefix+suffix,focalPlaneEle); 

            std::vector<double> ga_rs;
            std::vector<double> ele_rs;
            if(focalPlaneGa.size() >= 1000)
            {
                for(long unsigned int tmpIdx = 0;tmpIdx < focalPlaneGa.size();tmpIdx++)
                {
                    double gaX2 = focalPlaneGa[tmpIdx].x * focalPlaneGa[tmpIdx].x;
                    double gaY2 = focalPlaneGa[tmpIdx].y * focalPlaneGa[tmpIdx].y;
                    double gaR = sqrt(gaX2 + gaY2);
                    ga_rs.push_back(gaR);
                }
                std::sort(ga_rs.begin(),ga_rs.end());
                int dIdx = (int)(ga_rs.size() * 0.88) - 1;
                std::cout << "The d1288 value of Ga+:"<<ga_rs[dIdx] * 1e9 << "nm" << std::endl;

            }
            if(focalPlaneEle.size() >= 1000)//电子d1288
            {
                for(long unsigned int tmpIdx = 0;tmpIdx < focalPlaneEle.size();tmpIdx++)
                {
                    double eleX2 = focalPlaneEle[tmpIdx].x * focalPlaneEle[tmpIdx].x;
                    double eleY2 = focalPlaneEle[tmpIdx].y * focalPlaneEle[tmpIdx].y;
                    //double eleR = sqrt(eleX2 + eleY2);
                    double eleR = sqrt(eleY2);
                    ele_rs.push_back(eleR);
                }
                std::sort(ele_rs.begin(),ele_rs.end());
                int dIdx = (int)(ele_rs.size() * 0.88) - 1;
                std::cout << "The d1288 value of ele:"<<ele_rs[dIdx] * 1e9 << "nm" << std::endl;
            }



        }

        //在i为10的整数倍，判断是否需要添加ga离子
        //if(i % 10 == 0)
        if(true)
        {
            // //ga离子判断
            if(gaTimes.front() <= i)
            {
                Ga tmpGa[1] = {gaData.front()};
                gaData.pop();
                gaTimes.pop();
                hipMemcpy(ga_gpu + gaNum, tmpGa, sizeof(Ga), hipMemcpyHostToDevice);
                gaNum += 1;
            }
            //ele离子判断
            if(eleTimes.front() <= i)
            {
                Ele tmpEle[1] = {eleData.front()};
                eleData.pop();
                eleTimes.pop();
                hipMemcpy(ele_gpu + eleNum,tmpEle,sizeof(Ele),hipMemcpyHostToDevice);
                eleNum += 1;
            }
        }

        size_t threadsPerBlock = BLOCK_SIZE;

        //size_t threadsPerBlockGa = N_p_ga;
        //size_t threadsPerBlockEle = N_p_ele;
        size_t numberOfBlocksGa = (gaNum + threadsPerBlock - 1)/threadsPerBlock;
        size_t numberOfBlocksEle =(eleNum+ threadsPerBlock - 1)/threadsPerBlock;
        coulombGaForce<<<numberOfBlocksGa,threadsPerBlock>>>(ga_gpu,ele_gpu, dt,gaNum,eleNum);
        coulombEleForce<<<numberOfBlocksEle,threadsPerBlock>>>(ga_gpu,ele_gpu, dt,gaNum,eleNum);
        integrate_ga_position<<<numberOfBlocksGa,threadsPerBlock>>>(ga_gpu,dt,gaNum,S2 * 1e-3);
        integrate_ele_position<<<numberOfBlocksEle,threadsPerBlock>>>(ele_gpu,dt,eleNum,S2 * 1e-3);      
        //判断到达焦平面
        //if(i % focalStep == 0 && i >= focalStart)
        //if(i % (focalStep * 10) == 0)
        if(i % focalStep == 0)
        {
            //首先是数据拷贝，GPU->CPU
            int bytes_ga = gaNum * sizeof(Ga);
            int bytes_ele = eleNum * sizeof(Ele);
            gaBuff = std::vector<Ga>(gaNum);
            eleBuff = std::vector<Ele>(eleNum);
            hipMemcpy(gaBuff.data(),ga_gpu,bytes_ga,hipMemcpyDeviceToHost);
            hipMemcpy(eleBuff.data(),ele_gpu,bytes_ele,hipMemcpyDeviceToHost);

            //然后是拆分运行中粒子与到达焦平面粒子
            std::vector<Ga> tmpGaBuff;
            std::vector<Ele> tmpEleBuff;
            for(size_t j=0;j<gaBuff.size();j++)
            {
                if(gaBuff[j].z >= S2 * 1e-3)
                {
                    focalPlaneGa.push_back(gaBuff[j]);
                }
                else
                {
                    tmpGaBuff.push_back(gaBuff[j]);
                }
            }
            for(size_t j=0;j<eleBuff.size();j++)
            {
                if(eleBuff[j].z < S2 * 1e-3)
                {
                    tmpEleBuff.push_back(eleBuff[j]);
                }
                else
                {
                    focalPlaneEle.push_back(eleBuff[j]);
                }
            }
            gaBuff = tmpGaBuff;
            eleBuff = tmpEleBuff;
            //hipFree(ga_gpu);
            //hipFree(ele_gpu);
            gaNum = gaBuff.size();
            eleNum = eleBuff.size();
            bytes_ga = gaNum * sizeof(Ga);
            bytes_ele = eleNum * sizeof(Ele);
            //hipMallocManaged(&ga_gpu, bytes_ga);
            //hipMallocManaged(&ele_gpu, bytes_ele);
            hipMemcpy(ga_gpu, gaBuff.data(), bytes_ga, hipMemcpyHostToDevice);
            hipMemcpy(ele_gpu,eleBuff.data(),bytes_ele,hipMemcpyHostToDevice);
            // hipError_t err = hipGetLastError();
            // if (err != hipSuccess) {
            //     printf("CUDA Error: %s\n", hipGetErrorString(err));
            //     // Possibly: exit(-1) if program cannot continue....
            // }

        }
        //if((i+1)%gaStep == 0 || (i+1)%eleStep ==0)
        //{
        //    hipDeviceSynchronize();
        //}
    } 

    //保存位置文件
    std::cout << "Final output:" << gaBuff.size() << " "<< focalPlaneGa.size() << " "<< eleBuff.size() << " " << focalPlaneEle.size() << std::endl;
    //writeFile(positionPathGa,gaBuff);
    writeFile<Ga>(positionPathGa,gaBuff);
    writeFile<Ga>(focalPlanePathGa,focalPlaneGa);
    writeFile<Ele>(positionPathEle,eleBuff);
    writeFile<Ele>(focalPlanePathEle,focalPlaneEle);

    return 0;
}
